#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define BLOCKS   3
#define DATASIZE BLOCKS * 512
#define CHUNKNUM 1
#define N		 100


hipError_t square(int *result, int *data, int chunknum);

__global__ void squareKernel(int *result, int *data) {
	int i = threadIdx.x;
	result[i] = 0;
	result[i] = data[i] * data[i];
}

void deviceReset() {
// cudaDeviceReset must be called before exiting in order for profiling and
// tracing tools such as Parallel Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus;
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		//return 1;
	}

}

void setInput(int *data) {
	// Generate input data
	for (int i = 0; i < DATASIZE; i++) {
		data[i] = i;
	}
}

void printArray(char *content, int *input) {
	printf("%s\n", content);
	// Print the result array
	for (int i = 0; i < DATASIZE; i++)
		printf("i%d=%d, ", i, input[i]);
	printf("\n");
}

void dummyCudaMalloc(int **dummy_ptr) {
	hipError_t cudaStatus = hipMalloc((void**) dummy_ptr, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
}

int main() {
	int data[DATASIZE];
	int result[DATASIZE] = { 0 };
	int *dummy_ptr = 0;
	dummyCudaMalloc(&dummy_ptr);
	// Set false value in result array
	memset(result, 0, DATASIZE);
	setInput(data);

	// Print the input character
	// printArray("Input", data);
	// Search keyword in parallel.
	printf("square\n");

	hipError_t cudaStatus = square(result, data, CHUNKNUM);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// printArray("Result", result);

	hipFree(dummy_ptr);
	deviceReset();
	return 0;
}

// Helper function for using CUDA to search a list of characters in parallel.
hipError_t square(int *result, int *data, int num_kernel) {
	int *dev_data = 0;
	int *dev_result = 0;
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr,
				"hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
		hipFree(dev_result);
		return cudaStatus;
	}

	clock_t t = clock();
// Launch a search keyword kernel on the GPU with one thread for each element.
	for (int n = 0; n < N; n++) {
		for (int i = 0; i < num_kernel; i++) {
			int chunk_size = DATASIZE / num_kernel;
			// Allocate GPU buffers for result set.
			cudaStatus = hipMalloc((void**) &dev_result, chunk_size * sizeof(int));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			// Allocate GPU buffers for data set.
			cudaStatus = hipMalloc((void**) &dev_data, chunk_size * sizeof(int));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			// Copy input data from host memory to GPU buffers.
			cudaStatus = hipMemcpy(dev_data, data + i * chunk_size,
					chunk_size * sizeof(int), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
				goto Error;
			}

			squareKernel<<<BLOCKS, chunk_size>>>(dev_result, dev_data);

			// cudaDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr,
						"hipDeviceSynchronize returned error code %d after launching addKernel!\n",
						cudaStatus);
				goto Error;
			}
			// Copy result from GPU buffer to host memory.
			cudaStatus = hipMemcpy(result + i * chunk_size, dev_result,
					chunk_size * sizeof(int), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
				goto Error;
			}

			hipFree(dev_data);
			hipFree(dev_result);
			dev_data = NULL;
			dev_result = NULL;
		}
	}

	t = clock() - t;
	printf("%d kernel time for %d kernel(s): %f milliseconds\n",
			N, CHUNKNUM,
			((float) t) * 1000.0 / CLOCKS_PER_SEC);
	Error: hipFree(dev_result);
	return cudaStatus;

}
